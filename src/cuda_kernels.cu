#include "hip/hip_runtime.h"
// cuda_kernels.cu
#include <hip/hip_runtime.h>
#include "test_cuda_package/cuda_kernels.h"
#include <stdio.h>

__global__ void simple_cuda_kernel(float* data, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        data[idx] = idx + 2.0; // 예시 연산

        printf("Thread %d processed data[%d] = %f\n", idx, idx, data[idx]);
    }
}

void launch_simple_cuda_kernel(float* device_data, int size) {
    int threadsPerBlock = 256;
    int blocks = (size + threadsPerBlock - 1) / threadsPerBlock;
    simple_cuda_kernel<<<blocks, threadsPerBlock>>>(device_data, size);
    hipDeviceSynchronize(); // 커널 실행 완료 대기
}
